

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <float.h>

__global__ void transformKernel(float *x ,float *y,float *z,float *transform)
{
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	
	/*if (_finite(x[i])||
		_finite(y[i])||
		_finite(z[i]))
		return;*/
	 
	 
	 float x_,y_,z_;
	 x_ = static_cast<float> (transform [0] * x[i] + transform [1] * y[i] + transform [2] * z[i] + transform [3]);
	 y_ = static_cast<float> (transform [4] * x[i] + transform [5] * y[i] + transform [6] * z[i] + transform [7]);
     z_ = static_cast<float> (transform [8] * x[i] + transform [9] * y[i] + transform [10] * z[i] + transform [11]);
	 x[i]=x_;
	 y[i]=y_;
	 z[i]=z_;


}

void transformCloudGPU_CU(float *x ,float*y ,float *z,float *transform,int size)
{

	float *dev_x,*dev_y,*dev_z,*dev_transform;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return;
    }

	cudaStatus = hipMalloc((void**)&dev_x, size * sizeof(float));

	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return;
    }

    cudaStatus = hipMalloc((void**)&dev_y, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return;
    }

    cudaStatus = hipMalloc((void**)&dev_z, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return;
    }

	cudaStatus = hipMalloc((void**)&dev_transform, 16 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return;
    }


	cudaStatus = hipMemcpy(dev_x, x, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return;
    }

    cudaStatus = hipMemcpy(dev_y, y, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return;
    }

	cudaStatus = hipMemcpy(dev_z, z, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return;
    }

	cudaStatus = hipMemcpy(dev_transform, transform, 16 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return;
    }

	transformKernel<<<640,480>>>(dev_x,dev_y,dev_z,dev_transform);
	
	cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return;
    }

	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return;
    }

	cudaStatus = hipMemcpy(x,dev_x, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return;
    }

    cudaStatus = hipMemcpy( y,dev_y, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return;
    }

	cudaStatus = hipMemcpy( z, dev_z,size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return;
    }

	/*cudaStatus = cudaDeviceReset();
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaDeviceReset failed!");
        return ;
    }*/



}
